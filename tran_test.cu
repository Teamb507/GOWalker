
#include <stdio.h>
#include<stdint.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include<chrono>
int main(int argc, const char* argv[])
{
    uint32_t **data=(uint32_t **)malloc(512*8*sizeof(uint32_t *));
    for(int i=0;i<512*8;i++)
    {
        hipHostAlloc((void **)&data[i],4096*sizeof(uint32_t),hipHostMallocMapped);
    }
    for(int i=0;i<512*8;i++)
    {
        for(int j=0;j<4096;j++)
        {
            data[i][j]=i*4096+j;
        }
    }
    uint32_t *d_data;
    hipMalloc((void **)&d_data,512*4096*8*sizeof(uint32_t));
    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();
    for(int i=0;i<512*8;i++)
    {
        hipMemcpy(d_data+i*4096,data[i],4096*sizeof(uint32_t),hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    printf("elapsed time: %f s\n", elapsed_seconds.count());

    uint32_t *h_data;
    hipHostAlloc((void **)&h_data,512*4096*8*sizeof(uint32_t),hipHostMallocMapped);
    hipDeviceSynchronize();

    start = std::chrono::steady_clock::now();
    for(int i=0;i<512*8;i++)
    {
        for(int j=0;j<4096;j++)
        {
            
            h_data[i*4096+j]=data[i][j];
        }
    }
    hipMemcpy(d_data,h_data,512*4096*8*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    elapsed_seconds = end - start;
    printf("elapsed time: %f s\n", elapsed_seconds.count());
    return 0;
}